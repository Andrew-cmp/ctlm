#include <hip/hip_runtime.h>
#include <stdio.h>
#define CHECK(call)\
{\
  const hipError_t error=call;\
  if(error!=hipSuccess)\
  {\
      printf("ERROR: %s:%d,",__FILE__,__LINE__);\
      printf("code:%d,reason:%s\n",error,hipGetErrorString(error));\
      exit(1);\
  }\
}
__global__ void testkernel(){
    int ix=threadIdx.x+blockIdx.x*blockDim.x;
    int iy=threadIdx.y+blockIdx.y*blockDim.y;
    int iz=threadIdx.z+blockIdx.z*blockDim.z;
    printf("thread_id(%d,%d,%d) block_id(%d,%d,%d) coordinate(%d,%d,%d)\n",threadIdx.x,threadIdx.y,threadIdx.z,
            blockIdx.x,blockIdx.y,blockIdx.z,ix,iy,iz);
}

int main(){



    int device = 0;
    hipSetDevice(device);
    dim3 grid(1,1,1);
    dim3 block(256,2,2);
    testkernel<<<grid,block>>>();
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
        // Possibly: exit(-1) if program cannot continue....
    } 
    else{
        printf("1");
    }
    

}