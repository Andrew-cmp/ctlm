#include <hip/hip_runtime.h>
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <stdio.h>

int main(){
    int device = 0;
    hipDeviceProp_t prop;
    const Int CONTEXT_POOL_SIZE = 4;
    hipCtx_t contextPool[CONTEXT_POOL_SIZE];
    int smCounts[CONTEXT_POOL_SIZE];
    hipSetDevice(device);
    hipGetDeviceProperties(&prop, device);
    smCounts[0] = 1; smCounts[1] = 2;
    smCounts[3] = (prop. multiProcessorCount - 3) ∕ 3;
    smCounts[4] = (prop. multiProcessorCount - 3) ∕ 3 * 2;
    for (int i = 0; i < CONTEXT_POOL_SIZE; i++) {
        CUexecAffinityParam affinity;
        affinity.type = CU_EXEC_AFFINITY_TYPE_SM_COUNT;
        affinity.param.smCount.val = smCounts[i];
        cuCtxCreate_v3(&contextPool[i], affinity, 1, 0, deviceOrdinal);
    }
    for (int i = 0; i < CONTEXT_POOL_SIZE; i++) {
        std::thread([i]() {
        int numSms = 0;
        int numBlocksPerSm = 0;
        int numThreads = 128;
        CUexecAffinityParam affinity;
        hipCtxSetCurrent(contextPool[i]);
        cuCtxGetExecAffinity(&affinity, CU_EXEC_AFFINITY_TYPE_SM_COUNT);
        numSms = affinity.param.smCount.val;
        hipOccupancyMaxActiveBlocksPerMultiprocessor(
        &numBlocksPerSm, kernel, numThreads, 0);
        void *kernelArgs[] = {  };  // add kernel args 
        dim3 dimBlock(numThreads, 1, 1);
        dim3 dimGrid(numSms * numBlocksPerSm, 1, 1);
        hipLaunchCooperativeKernel((void*)my_kernel, dimGrid, dimBlock, kernelArgs);
        };
    )
    }
}

