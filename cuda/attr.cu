#include <hip/hip_runtime.h>
#include <stdio.h>

#define GETATTR(func, device) \
    int func##device;\
    hipDeviceGetAttribute(&func##device, func, device);\
    printf(#func":%d \n",func##device);

int main() {

    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    printf("系统检测到 %d 个CUDA设备\n", deviceCount);

    #pragma unroll
    for (int dev = 0; dev < deviceCount; dev++){
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, dev);
        printf("Device name: %s\n" , prop.name) ;
        printf("Compute capability: %d\n"  , prop.minor) ;
        GETATTR(hipDeviceAttributeMultiprocessorCount,dev);
        hipError_t err = hipDeviceSetCacheConfig(hipFuncCachePreferShared);
        if (err != hipSuccess) {
           printf("Error setting shared memory config: %s",hipGetErrorString(err)) ;
        }   hipFuncCache_t pCacheConfig;
        hipError_t err3 = hipDeviceGetCacheConfig(&pCacheConfig);
        if (err != hipSuccess) {
            printf("Error setting shared memory config: %s",hipGetErrorString(err3)) ;
        }
        printf("sharedMemPerMultiprocessor:%d\n", prop.sharedMemPerMultiprocessor );
        GETATTR(hipDeviceAttributeMaxSharedMemoryPerBlock,dev);
        GETATTR(hipDeviceAttributeWarpSize,dev);
        GETATTR(hipDeviceAttributeComputeCapabilityMajor,dev);
        GETATTR(hipDeviceAttributeMaxBlockDimX,dev);
        GETATTR(hipDeviceAttributeMaxBlockDimY,dev);
        GETATTR(hipDeviceAttributeMaxBlockDimZ,dev);
        GETATTR(hipDeviceAttributeMaxGridDimX,dev);
        GETATTR(hipDeviceAttributeMaxGridDimY,dev);
        GETATTR(hipDeviceAttributeMaxGridDimZ,dev);
        GETATTR(hipDeviceAttributeTotalConstantMemory,dev);
        GETATTR(hipDeviceAttributeMaxPitch,dev);
        GETATTR(hipDeviceAttributeClockRate,dev);
        GETATTR(hipDeviceAttributeMemoryBusWidth,dev);

    }




}
