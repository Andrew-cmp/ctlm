#include <hip/hip_runtime.h>
#include <stdio.h>

#define GETATTR(func, device) \
    int func##device;\
    hipDeviceGetAttribute(&func##device, func, device);\
    printf(#func":%d \n",func##device);

int main() {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    printf("Device name: %s\n" , prop.name) ;
    printf("Compute capability: %d\n"  , prop.minor) ;
    

    GETATTR(hipDeviceAttributeMultiprocessorCount,0);
    hipError_t err = hipDeviceSetCacheConfig(hipFuncCachePreferShared);
    if (err != hipSuccess) {
       printf("Error setting shared memory config: %s",hipGetErrorString(err)) ;
    }   hipFuncCache_t pCacheConfig;
    hipError_t err3 = hipDeviceGetCacheConfig(&pCacheConfig);
    if (err != hipSuccess) {
        printf("Error setting shared memory config: %s",hipGetErrorString(err3)) ;
    }
    printf("%d\n",pCacheConfig);

    printf("sharedMemPerMultiprocessor:%d\n", prop.sharedMemPerMultiprocessor );
    GETATTR(hipDeviceAttributeMaxSharedMemoryPerBlock,0);
    GETATTR(hipDeviceAttributeWarpSize,0);
    GETATTR(hipDeviceAttributeComputeCapabilityMajor,0);
    GETATTR(hipDeviceAttributeMaxBlockDimX,0);
    GETATTR(hipDeviceAttributeMaxBlockDimY,0);
    GETATTR(hipDeviceAttributeMaxBlockDimZ,0);
    GETATTR(hipDeviceAttributeMaxGridDimX,0);
    GETATTR(hipDeviceAttributeMaxGridDimY,0);
    GETATTR(hipDeviceAttributeMaxGridDimZ,0);
    GETATTR(hipDeviceAttributeTotalConstantMemory,0);
    GETATTR(hipDeviceAttributeMaxPitch,0);
}