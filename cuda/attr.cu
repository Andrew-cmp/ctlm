#include <hip/hip_runtime.h>
#include <stdio.h>

#define GETATTR(func, device) \
    int func##device;\
    hipDeviceGetAttribute(&func##device, func, device);\
    printf(#func":%d \n",func##device);

int main() {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    printf("Device name: %s\n" , prop.name) ;
    printf("Compute capability: %d\n"  , prop.minor) ;
    
    hipError_t err = hipDeviceSetCacheConfig(hipFuncCachePreferEqual);

    GETATTR(hipDeviceAttributeMultiprocessorCount,0);
    GETATTR(hipDeviceAttributeMaxSharedMemoryPerBlock,0);
    GETATTR(hipDeviceAttributeWarpSize,0);
    GETATTR(hipDeviceAttributeComputeCapabilityMajor,0);
    GETATTR(hipDeviceAttributeMaxBlockDimX,0);
    GETATTR(hipDeviceAttributeMaxBlockDimY,0);
    GETATTR(hipDeviceAttributeMaxBlockDimZ,0);
    GETATTR(hipDeviceAttributeMaxGridDimX,0);
    GETATTR(hipDeviceAttributeMaxGridDimY,0);
    GETATTR(hipDeviceAttributeMaxGridDimZ,0);
    GETATTR(hipDeviceAttributeTotalConstantMemory,0);
    GETATTR(hipDeviceAttributeMaxPitch,0);
}