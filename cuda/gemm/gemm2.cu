#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>
#define ELE_TYPE float
#define TILE_M 2
#define TILE_N 2
template<uint32_t M,uint32_t N,uint32_t K>
__global__ void gemm_kernel(ELE_TYPE* A, ELE_TYPE* B,ELE_TYPE* C){
    // row指的是行上的坐标，而不是指的是第几行
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    // col指的是列上的坐标，而不是第几列
    int y = blockIdx.y*blockDim.y + threadIdx.y;

    __shared__ ELE_TYPE Sa[M/TILE_M][K];
    __shared__ ELE_TYPE Sb[K][N/TILE_N];
    ELE_TYPE sum = 0;
    //先从global mem移动到share mem
    __syncthreads();
    for(int i = threadIdx.x;i < K;i+=TILE_M){
        Sa[threadIdx.y][i] = A[y*K+i];
    } 
    for(int i = threadIdx.y;i < K;i += TILE_N){
        Sb[i][threadIdx.x] = B[i*N+x];
    }
    __syncthreads();
    for(int s = 0;s < K;s++ ){
        sum += Sa[threadIdx.y][s] * Sb[s][threadIdx.x];
    }
    C[y*N+x] = sum;

    // __syncthreads();
    // for(int i = threadIdx.y;i < K;i+=TILE_M){
    //     Sa[threadIdx.x][i] = A[x*K+i];
    // } 
    // for(int i = threadIdx.x;i < K;i += TILE_N){
    //     Sb[i][threadIdx.y] = B[i*N+y];
    // }
    // __syncthreads();
    // for(int s = 0;s < K;s++ ){
    //     sum += Sa[threadIdx.x][s] * Sb[s][threadIdx.y];
    // }
    // C[x*N+y] = sum;
    
}

int main(){

    ///这个不加还不能当cuda模板参数
    ///要求必须是编译时期已知且运行时不会变的constant
    constexpr uint32_t N = 16;
    constexpr uint32_t M = 32;
    constexpr uint32_t K = 8;
    int size_A = M*K*sizeof(ELE_TYPE);
    int size_B = K*N*sizeof(ELE_TYPE);
    int size_C = M*N*sizeof(ELE_TYPE);
    ELE_TYPE * h_a =(ELE_TYPE*)malloc(size_A);
    ELE_TYPE * h_b =(ELE_TYPE*)malloc(size_B);
    ELE_TYPE * h_c =(ELE_TYPE*)malloc(size_C);
    for (int i = 0; i < M * K; ++i) h_a[i] = 2.0f;
    for (int i = 0; i < K * N; ++i) h_b[i] = 2.0f;
    ELE_TYPE *d_a, *d_b, *d_c;
    hipMalloc(&d_a,M*K*sizeof(ELE_TYPE));
    hipMalloc(&d_b,K*N*sizeof(ELE_TYPE));
    hipMalloc(&d_c,M*N*sizeof(ELE_TYPE));
    
    hipMemcpy(d_a,h_a,size_A,hipMemcpyHostToDevice);
    hipMemcpy(d_b,h_b,size_B,hipMemcpyHostToDevice);
    
    dim3 blockDim(4,4);
    dim3 gridDim((N+threadperblock.x-1)/threadperblock.x,
                      (M+threadperblock.y-1)/threadperblock.y );
    //草，大模型给的代码，下面的GridDim和blockDim位置对调了
    //gemm_kernel<M,N,K><<<blockDim,gridDim>>>(d_a,d_b,d_c);
    gemm_kernel<M,N,K><<<gridDim,blockDim>>>(d_a,d_b,d_c);

    hipMemcpy(h_c,d_c,size_C,hipMemcpyDeviceToHost);

    for(int i = 0;i < M;++i){
        for(int j = 0;j < N;j++){
            printf("%0.1f ",*(h_c+i*N+j));
        }
        printf("\n");
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(h_a);
    free(h_b);
    free(h_c);
    
}