#include <iostream>
#include <hip/hip_runtime.h>

#define N1 64
#define N2 512
#define N3 512
#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void main_kernel0(float* __restrict__ p0, float* __restrict__ p1, float* __restrict__ T_batch_matmul_NN) {
  float T_batch_matmul_NN_local[2048];
  __shared__ float p0_shared[8];
  __shared__ float p1_shared[256];
  for (int k_0 = 0; k_0 < 512; ++k_0) {
    __syncthreads();
    for (int ax0_ax1_ax2_fused = 0; ax0_ax1_ax2_fused < 8; ++ax0_ax1_ax2_fused) {
      p0_shared[ax0_ax1_ax2_fused] = p0[((((((int)blockIdx.x) >> 1) * 4096) + (ax0_ax1_ax2_fused * 512)) + k_0)];
    }
    for (int ax0_ax1_ax2_fused_1 = 0; ax0_ax1_ax2_fused_1 < 256; ++ax0_ax1_ax2_fused_1) {
      p1_shared[ax0_ax1_ax2_fused_1] = p1[(((((((int)blockIdx.x) >> 4) * 262144) + (k_0 * 512)) + ((((int)blockIdx.x) & 1) * 256)) + ax0_ax1_ax2_fused_1)];
    }
    __syncthreads();
    for (int i_3 = 0; i_3 < 2; ++i_3) {
      for (int j_3 = 0; j_3 < 2; ++j_3) {
        for (int i_4 = 0; i_4 < 4; ++i_4) {
          for (int j_4 = 0; j_4 < 32; ++j_4) {
            if (k_0 == 0) {
              T_batch_matmul_NN_local[((((i_3 * 256) + (i_4 * 64)) + (j_3 * 32)) + j_4)] = 0.000000e+00f;
              T_batch_matmul_NN_local[(((((i_3 * 256) + (i_4 * 64)) + (j_3 * 32)) + j_4) + 512)] = 0.000000e+00f;
              T_batch_matmul_NN_local[(((((i_3 * 256) + (i_4 * 64)) + (j_3 * 32)) + j_4) + 1024)] = 0.000000e+00f;
              T_batch_matmul_NN_local[(((((i_3 * 256) + (i_4 * 64)) + (j_3 * 32)) + j_4) + 1536)] = 0.000000e+00f;
            }
            T_batch_matmul_NN_local[((((i_3 * 256) + (i_4 * 64)) + (j_3 * 32)) + j_4)] = (T_batch_matmul_NN_local[((((i_3 * 256) + (i_4 * 64)) + (j_3 * 32)) + j_4)] + (p0_shared[((i_3 * 4) + i_4)] * p1_shared[((j_3 * 32) + j_4)]));
            T_batch_matmul_NN_local[(((((i_3 * 256) + (i_4 * 64)) + (j_3 * 32)) + j_4) + 512)] = (T_batch_matmul_NN_local[(((((i_3 * 256) + (i_4 * 64)) + (j_3 * 32)) + j_4) + 512)] + (p0_shared[((i_3 * 4) + i_4)] * p1_shared[(((j_3 * 32) + j_4) + 64)]));
            T_batch_matmul_NN_local[(((((i_3 * 256) + (i_4 * 64)) + (j_3 * 32)) + j_4) + 1024)] = (T_batch_matmul_NN_local[(((((i_3 * 256) + (i_4 * 64)) + (j_3 * 32)) + j_4) + 1024)] + (p0_shared[((i_3 * 4) + i_4)] * p1_shared[(((j_3 * 32) + j_4) + 128)]));
            T_batch_matmul_NN_local[(((((i_3 * 256) + (i_4 * 64)) + (j_3 * 32)) + j_4) + 1536)] = (T_batch_matmul_NN_local[(((((i_3 * 256) + (i_4 * 64)) + (j_3 * 32)) + j_4) + 1536)] + (p0_shared[((i_3 * 4) + i_4)] * p1_shared[(((j_3 * 32) + j_4) + 192)]));
          }
        }
      }
    }
  }
  for (int ax1 = 0; ax1 < 8; ++ax1) {
    for (int ax2 = 0; ax2 < 64; ++ax2) {
      T_batch_matmul_NN[(((((((int)blockIdx.x) >> 1) * 4096) + (ax1 * 512)) + ((((int)blockIdx.x) & 1) * 256)) + ax2)] = T_batch_matmul_NN_local[((ax1 * 64) + ax2)];
      T_batch_matmul_NN[((((((((int)blockIdx.x) >> 1) * 4096) + (ax1 * 512)) + ((((int)blockIdx.x) & 1) * 256)) + ax2) + 64)] = T_batch_matmul_NN_local[(((ax1 * 64) + ax2) + 512)];
      T_batch_matmul_NN[((((((((int)blockIdx.x) >> 1) * 4096) + (ax1 * 512)) + ((((int)blockIdx.x) & 1) * 256)) + ax2) + 128)] = T_batch_matmul_NN_local[(((ax1 * 64) + ax2) + 1024)];
      T_batch_matmul_NN[((((((((int)blockIdx.x) >> 1) * 4096) + (ax1 * 512)) + ((((int)blockIdx.x) & 1) * 256)) + ax2) + 192)] = T_batch_matmul_NN_local[(((ax1 * 64) + ax2) + 1536)];
    }
  }
}

int main() {
  // 定义矩阵维度
  int size_A = 2 * N1 * N2;  // 2 * 64 * 512
  int size_B = 2 * N2 * N3;  // 2 * 512 * 512
  int size_C = 2 * N1 * N2;  // 2 * 64 * 512

  // 分配内存
  float *h_A = new float[size_A];
  float *h_B = new float[size_B];
  float *h_C = new float[size_C];

  // 填充矩阵 A 和 B，假设是随机初始化或某种已知值
  for (int i = 0; i < size_A; ++i) h_A[i] = static_cast<float>(rand() % 100);
  for (int i = 0; i < size_B; ++i) h_B[i] = static_cast<float>(rand() % 100);

  // 设备内存分配
  float *d_A, *d_B, *d_C;
  hipMalloc((void**)&d_A, size_A * sizeof(float));
  hipMalloc((void**)&d_B, size_B * sizeof(float));
  hipMalloc((void**)&d_C, size_C * sizeof(float));

  // 将数据从 Host 复制到 Device
  hipMemcpy(d_A, h_A, size_A * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, size_B * sizeof(float), hipMemcpyHostToDevice);

  // 设置块和网格大小
  dim3 blockSize(16, 16);
  dim3 gridSize((N1 + blockSize.x - 1) / blockSize.x, (N2 + blockSize.y - 1) / blockSize.y);

  // 调用核函数
  main_kernel0<<<gridSize, blockSize>>>(d_A, d_B, d_C);

  // 检查是否有任何 CUDA 错误
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
      std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
      return -1;
  }

  // 从 Device 复制结果到 Host
  hipMemcpy(h_C, d_C, size_C * sizeof(float), hipMemcpyDeviceToHost);

  // 输出部分结果（示例）
  for (int i = 0; i < N1; ++i) {
      for (int j = 0; j < N2; ++j) {
          std::cout << h_C[i * N2 + j] << " ";
      }
      std::cout << std::endl;
  }

  // 释放内存
  delete[] h_A;
  delete[] h_B;
  delete[] h_C;
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);

  return 0;
}