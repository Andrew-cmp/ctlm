#include <hip/hip_runtime.h>
#include <iostream>


const char* hipDrvGetErrorString(hipError_t err) {
    const char* errorStr;
    hipDrvGetErrorString(err, &errorStr);  // 获取错误字符串
    return errorStr;
}
void gemm_cuda(float* A, float* B, float* C, int M, int N, int K) {
    // CUDA 初始化
    hipError_t err = hipSetDevice(0);
    if (err != hipSuccess) {
        std::cerr << "CUDA set device failed: " << hipGetErrorString(err) << std::endl;
        return;
    }

    // 计算 grid 和 block 尺寸
    dim3 block(16, 16);  // 线程块大小 16x16
    dim3 grid((N + block.x - 1) / block.x, (M + block.y - 1) / block.y);

    // 分配设备内存
    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, M * K * sizeof(float));
    hipMalloc((void**)&d_B, K * N * sizeof(float));
    hipMalloc((void**)&d_C, M * N * sizeof(float));

    // 将数据从主机传输到设备
    hipMemcpy(d_A, A, M * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, K * N * sizeof(float), hipMemcpyHostToDevice);

    // 加载内核函数
    hipFunction_t gemm_func;
    hipModule_t module;
    hipError_t res = hipModuleLoad(&module, "gemm.ptx"); // 加载 PTX 文件
    if (res != hipSuccess) {
        std::cerr << "Failed to load module" << std::endl;
        return;
    }

    // 获取内核函数
    res = hipModuleGetFunction(&gemm_func, module, "_Z11gemm_kernelPfS_S_");
    if (res != hipSuccess) {
        std::cerr << "Failed to get function" << std::endl;
        std::cerr <<hipDrvGetErrorString(res)<< std::endl;
        return;
    }

    // 设置内核参数
    void* kernel_params[] = {
        (void*)&d_A,
        (void*)&d_B,
        (void*)&d_C,
        (void*)&M,
        (void*)&N,
        (void*)&K
    };

    // 启动内核
    res = hipModuleLaunchKernel(gemm_func,
                         grid.x, grid.y, 1,        // Grid size
                         block.x, block.y, 1,      // Block size
                         0,                         // Shared memory size
                         0,                         // Stream
                         kernel_params, nullptr);  // Parameters

    if (res != hipSuccess) {
        std::cerr << "Failed to launch kernel" << std::endl;
        return;
    }

    // 等待内核完成
    hipDeviceSynchronize();

    // 将结果从设备复制回主机
    hipMemcpy(C, d_C, M * N * sizeof(float), hipMemcpyDeviceToHost);

    // 清理资源
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

int main() {
    int M = 1024, N = 1024, K = 1024;
    float *A = new float[M * K];
    float *B = new float[K * N];
    float *C = new float[M * N];

    // 初始化 A 和 B
    for (int i = 0; i < M * K; ++i) A[i] = 1.0f;
    for (int i = 0; i < K * N; ++i) B[i] = 1.0f;

    gemm_cuda(A, B, C, M, N, K);

    // 输出 C 的部分结果
    std::cout << "C[0][0]: " << C[0] << std::endl;

    // 清理主机内存
    delete[] A;
    delete[] B;
    delete[] C;

    return 0;
}
