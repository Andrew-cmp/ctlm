#include <iostream>
#include <hip/hip_runtime.h>

#define N 64   // A的行数
#define M 512  // A的列数，也是B的行数
#define K 512  // B的列数
__global__ void __maxnreg__(50) gemm_kernel(float* A, float* B, float* C) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < M && col < N) {
        float sum = 0.0f;
        for (int k = 0; k < K; k++) {
            sum += A[row * K + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}
int main() {
    // 主机端矩阵（初始化为一些数据）
    float *h_A = new float[N * M]; // 64x512
    float *h_B = new float[M * K]; // 512x512
    float *h_C = new float[N * K]; // 64x512

    // 初始化矩阵数据（可以根据需要填充数据）
    for (int i = 0; i < N * M; i++) {
        h_A[i] = static_cast<float>(i % 100);  // 仅为示例，实际应用中需要根据需要填充数据
    }

    for (int i = 0; i < M * K; i++) {
        h_B[i] = static_cast<float>((i + 1) % 100);  // 仅为示例
    }

    // 设备端矩阵指针
    float *d_A, *d_B, *d_C;
    
    // 分配设备内存
    hipMalloc((void**)&d_A, N * M * sizeof(float));
    hipMalloc((void**)&d_B, M * K * sizeof(float));
    hipMalloc((void**)&d_C, N * K * sizeof(float));

    // 将数据从主机复制到设备
    hipMemcpy(d_A, h_A, N * M * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, M * K * sizeof(float), hipMemcpyHostToDevice);

    // 设置 CUDA 核函数的网格和块的维度
    dim3 threadsPerBlock(16, 16); // 每个块中有 16x16 个线程
    dim3 numBlocks((K + 15) / 16, (N + 15) / 16); // 计算块的数量

    hipFuncSetCacheConfig(reinterpret_cast<const void*>(gemm_kernel), hipFuncCachePreferEqual);
    // 调用 CUDA 核函数进行矩阵乘法
    gemm_kernel<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C);

    // 检查核函数调用是否有错误
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
        return -1;
    }

    // 将结果从设备复制回主机
    hipMemcpy(h_C, d_C, N * K * sizeof(float), hipMemcpyDeviceToHost);

    // 打印结果（只打印部分数据以避免输出过多）
    std::cout << "Result (first 5 elements of C):\n";
    for (int i = 0; i < 5; i++) {
        std::cout << h_C[i] << " ";
    }
    std::cout << std::endl;

    // 释放设备内存
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // 释放主机内存
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;

    return 0;
}