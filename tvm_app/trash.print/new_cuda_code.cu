
#include <hip/hip_runtime.h>

#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __maxnreg__(1) main_kernel0(float* __restrict__ A, float* __restrict__ B) {
  __shared__ float A_shared[130];
  for (int ax0_0 = 0; ax0_0 < 2; ++ax0_0) {
    if (((ax0_0 * 64) + (((int)threadIdx.x) >> 1)) < 65) {
      A_shared[((ax0_0 * 128) + ((int)threadIdx.x))] = A[(((((int)blockIdx.x) * 128) + (ax0_0 * 128)) + ((int)threadIdx.x))];
    }
  }
  __syncthreads();
  B[((((int)blockIdx.x) * 128) + ((int)threadIdx.x))] = ((A_shared[((int)threadIdx.x)] + A_shared[(((int)threadIdx.x) + 1)]) + A_shared[(((int)threadIdx.x) + 2)]);

}

